#include "hip/hip_runtime.h"
#include <cstdlib>
#include <hipblas.h>
#include <iostream>
#include <time.h>
#include <tools/utils.h>

#include "queueDistLocks.cuh"
#include "queueShared.cuh"
#include "queuingPerProc.cuh"
#include "techniqueMegakernel.cuh"
#include "techniqueKernels.cuh"
#include "techniqueDynamicParallelism.cuh"
#include "segmentedStorage.cuh"

#include "procedureInterface.cuh"
#include "procinfoTemplate.cuh"
#include "random.cuh"

namespace Tools
{
	class CublasError : public std::runtime_error
	{
	private:
		static __host__ std::string genErrorString(hipblasStatus_t error, const char* file, int line)
		{
			std::string strerror;
			switch (error)
			{
			case HIPBLAS_STATUS_NOT_INITIALIZED :
				strerror = "HIPBLAS_STATUS_NOT_INITIALIZED";
				break;
			case HIPBLAS_STATUS_ALLOC_FAILED :
				strerror = "HIPBLAS_STATUS_ALLOC_FAILED";
				break;
			case HIPBLAS_STATUS_INVALID_VALUE :
				strerror = "HIPBLAS_STATUS_INVALID_VALUE";
				break;
			case HIPBLAS_STATUS_ARCH_MISMATCH :
				strerror = "HIPBLAS_STATUS_ARCH_MISMATCH";
				break;
			case HIPBLAS_STATUS_MAPPING_ERROR :
				strerror = "HIPBLAS_STATUS_MAPPING_ERROR";
				break;
			case HIPBLAS_STATUS_EXECUTION_FAILED :
				strerror = "HIPBLAS_STATUS_EXECUTION_FAILED";
				break;
			case HIPBLAS_STATUS_INTERNAL_ERROR :
				strerror = "HIPBLAS_STATUS_INTERNAL_ERROR";
				break;
			case HIPBLAS_STATUS_NOT_SUPPORTED :
				strerror = "HIPBLAS_STATUS_NOT_SUPPORTED";
				break;
			case HIPBLAS_STATUS_UNKNOWN :
				strerror = "HIPBLAS_STATUS_UNKNOWN";
				break;
			}
		
			return std::string(file) + '(' + std::to_string(static_cast<long long>(line)) + "): error: " + strerror;
		}
	public:
		__host__ CublasError(hipblasStatus_t error, const char* file, int line)
		: runtime_error(genErrorString(error, file, line))
		{
		}
	};

	inline __host__ void cublasError(hipblasStatus_t error, const char* file, int line)
	{
		if (error != HIPBLAS_STATUS_SUCCESS)
			throw CublasError(error, file, line);
	}
}

#define CUBLAS_CHECKED_CALL(call) Tools::cublasError(call, __FILE__, __LINE__)

struct dim2 { uint x, y; };

struct MatmulConfig
{
	float *A, *B, *C;
	size_t n;
	dim2 gridDim_;
};

__constant__ MatmulConfig config;

class MatmulTask : public ::Procedure
{
public:
	static const int NumThreads = BLOCK_SIZE * BLOCK_SIZE;
	static const bool ItemInput = false; // false results in a lvl 1	task
	static const int sharedMemory = 2 * sizeof(float) * NumThreads;	// shared memory requirements 
	
	typedef uint ExpectedData;

	template<class Q, class Context>
	static __device__ __inline__ void execute(int threadId, int numThreads, Q* queue, ExpectedData* ptaskid, volatile uint* shared) 
	{
		float*& A = config.A;
		float*& B = config.B;
		float*& C = config.C;
		size_t& n = config.n;
		dim2& gridDim_ = config.gridDim_;
	
		const uint taskid = *ptaskid;
	
		struct { uint x, y; } blockDim;
		blockDim.x = BLOCK_SIZE;
		blockDim.y = BLOCK_SIZE;
		
		struct { uint x, y; } blockIdx;
		blockIdx.x = taskid % gridDim_.x;
		blockIdx.y = taskid / gridDim_.x;
		
		struct { uint x, y; } threadIdx;
		threadIdx.x = threadId % BLOCK_SIZE;
		threadIdx.y = threadId / BLOCK_SIZE;

		float sum = 0.0f;

#ifndef MATMUL_USE_SHARED
		int ia = (blockDim.y * blockIdx.y + threadIdx.y) * n;
		int ib = blockDim.x * blockIdx.x + threadIdx.x;
		int ic = ia + ib;

		// Multiply two matrices
		for (int k = 0; k < n; k++)
			sum += A [ia + k] * B [ib + k * n];
#else
		// Base indexes inside A and B
		int ia = (blockDim.y * blockIdx.y) * n;
		int ib = blockDim.x * blockIdx.x;
	
		// Subindex inside a "tile"
		int tileidx = n * threadIdx.y + threadIdx.x;
	
		// Index in C
		int ic = ia + ib + tileidx;

		// Shared memory for the "tile" sub-matrix of A and B
		float* As = (float*)shared;
		float* Bs = (float*)shared + BLOCK_SIZE * BLOCK_SIZE;

		// Go through "tiles" of size blockDim.x * blockDim.y
		for (uint aoff = 0, boff = 0; aoff < n; aoff += blockDim.x, boff += blockDim.y * n)
		{
			// Load the "tile" matrices from global memory to shared memory
			As [threadIdx.y * BLOCK_SIZE + threadIdx.x] = A [ia + aoff + tileidx];
			Bs [threadIdx.y * BLOCK_SIZE + threadIdx.x] = B [ib + boff + tileidx];

			// Synchronize to make sure the matrices are loaded
			Context::sync();

			// Multiply the two matrices
			for (int k = 0; k < BLOCK_SIZE; k++)
				sum += As [threadIdx.y * BLOCK_SIZE + k] * Bs [k * BLOCK_SIZE + threadIdx.x];

			// Synchronize to make sure that the preceding
			// computation is done before loading two new
			// sub-matrices of A and B in the next iteration
			Context::sync();
		}
#endif
		// Write the block sub-matrix to global memory
		// each thread writes one element
		C [ic] = sum;
	}

	template<class Q>
	__device__ __inline__ static void init(Q* q, int id)
	{
		q->template enqueueInitial<MatmulTask>(id);
	}
};

enum MatmulVersion
{
	CUBLAS,
	CUDA,
	WHIPPLETREE
};

__global__ void cuda_matmul(float* A, float* B, float* C, size_t n)
{
    float sum = 0.0f;

#ifndef MATMUL_USE_SHARED
	int ia = (blockDim.y * blockIdx.y + threadIdx.y) * n;
	int ib = blockDim.x * blockIdx.x + threadIdx.x;
	int ic = ia + ib;

	// Multiply two matrices
	for (int k = 0; k < n; k++)
		sum += A [ia + k] * B [ib + k * n];
#else
    // Base indexes inside A and B
    int ia = (blockDim.y * blockIdx.y) * n;
    int ib = blockDim.x * blockIdx.x;
    
    // Subindex inside a "tile"
    int tileidx = n * threadIdx.y + threadIdx.x;
    
    // Index in C
    int ic = ia + ib + tileidx;

    int aoff = 0, boff = 0;

    // Shared memory for the "tile" sub-matrix of A and B
    __shared__ float As [BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs [BLOCK_SIZE][BLOCK_SIZE];

    // Go through "tiles" of size blockDim.x * blockDim.y
    for (; aoff < n; aoff += blockDim.x, boff += blockDim.y * n)
    {
        // Load the "tile" matrices from global memory to shared memory
        As [threadIdx.y][threadIdx.x] = A [ia + aoff + tileidx];
        Bs [threadIdx.y][threadIdx.x] = B [ib + boff + tileidx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices
        for (int k = 0; k < BLOCK_SIZE; k++)
            sum += As [threadIdx.y][k] * Bs [k][threadIdx.x];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }
#endif
    // Write the block sub-matrix to global memory
    // each thread writes one element
    C [ic] = sum;
}

class Matmul
{
public :
	//lets use a dist locks queue for each procedure, which can hold 12k elements
	template<class ProcInfo>
	class MyQueue : public PerProcedureQueueTyping<QueueDistLocksOpt_t, 96 * 1024, false>::Type<ProcInfo> { };

	//and lets use a Megakernel which can execute multiple workpackages concurrently (dynamic)
	//and offers a maximum of 16k shared memory
	typedef Megakernel::SimplePointed16336<MyQueue, ProcInfo<MatmulTask> > MyTechnique;

	Matmul(float* Ah, float* Bh, float* Ch, size_t n, MatmulVersion version, float* time = NULL)
	{
		MatmulConfig hconfig;
		float*& A = hconfig.A;
		float*& B = hconfig.B;
		float*& C = hconfig.C;
		hconfig.n = n;
		hconfig.gridDim_.x = n / BLOCK_SIZE;
		hconfig.gridDim_.y = n / BLOCK_SIZE;
	
		CUDA_CHECKED_CALL(hipMalloc(&A, sizeof(float) * n * n));
		CUDA_CHECKED_CALL(hipMalloc(&B, sizeof(float) * n * n));
		CUDA_CHECKED_CALL(hipMalloc(&C, sizeof(float) * n * n));

		CUDA_CHECKED_CALL(hipMemcpyToSymbol(HIP_SYMBOL(config), &hconfig, sizeof(MatmulConfig)));

		CUDA_CHECKED_CALL(hipMemcpy(A, Ah, sizeof(float) * n * n, hipMemcpyHostToDevice));
		CUDA_CHECKED_CALL(hipMemcpy(B, Bh, sizeof(float) * n * n, hipMemcpyHostToDevice));

		if (version == MatmulVersion::CUBLAS)
		{		
			hipblasHandle_t handle;
			CUBLAS_CHECKED_CALL(hipblasCreate(&handle));

			volatile struct timespec start;
			clock_gettime(CLOCK_REALTIME, (struct timespec*)&start);

			float fone = 1.0f, fzero = 0.0f;
			CUBLAS_CHECKED_CALL(hipblasSgemm(handle,
				hipblasOperation_t::HIPBLAS_OP_T, hipblasOperation_t::HIPBLAS_OP_T,
				n, n, n, &fone, A, n, B, n, &fzero, C, n));
			
			CUDA_CHECKED_CALL(hipDeviceSynchronize());

			volatile struct timespec finish;
			clock_gettime(CLOCK_REALTIME, (struct timespec*)&finish);

			hipblasDestroy(handle);
			
			if (time)
				*time = (float)((double)0.000000001 * (finish.tv_nsec - start.tv_nsec) +
					finish.tv_sec - start.tv_sec);

		}
		if (version == MatmulVersion::CUDA)
		{
			volatile struct timespec start;
			clock_gettime(CLOCK_REALTIME, (struct timespec*)&start);

		    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    		dim3 blocks( n / threads.x, n / threads.y);
			cuda_matmul<<<blocks, threads>>>(A, B, C, n);
			CUDA_CHECKED_CALL(hipGetLastError());
			CUDA_CHECKED_CALL(hipDeviceSynchronize());

			volatile struct timespec finish;
			clock_gettime(CLOCK_REALTIME, (struct timespec*)&finish);

			if (time)
				*time = (float)((double)0.000000001 * (finish.tv_nsec - start.tv_nsec) +
					finish.tv_sec - start.tv_sec);
		}
		if (version == MatmulVersion::WHIPPLETREE)
		{
			MyTechnique technique;
			technique.init();

			technique.insertIntoQueue<MatmulTask>(hconfig.gridDim_.x * hconfig.gridDim_.y);

			volatile struct timespec start;
			clock_gettime(CLOCK_REALTIME, (struct timespec*)&start);

			technique.execute(0);
			CUDA_CHECKED_CALL(hipDeviceSynchronize());

			volatile struct timespec finish;
			clock_gettime(CLOCK_REALTIME, (struct timespec*)&finish);

			if (time)
				*time = (float)((double)0.000000001 * (finish.tv_nsec - start.tv_nsec) +
					finish.tv_sec - start.tv_sec);
		}

		CUDA_CHECKED_CALL(hipMemcpy(Ch, C, sizeof(float) * n * n, hipMemcpyDeviceToHost));

		CUDA_CHECKED_CALL(hipFree(A));
		CUDA_CHECKED_CALL(hipFree(B));
		CUDA_CHECKED_CALL(hipFree(C));
	}
};

int main(int argc, char** argv)
{
	using namespace std;

	if (argc != 2)
	{
		cout << "Usage: " << argv[0] << " <n>" << endl;
		return 1;
	}

	int count;
	CUDA_CHECKED_CALL(hipGetDeviceCount(&count));
	if (!count)
	{
		cerr << "No CUDA devices available" << endl;
		return -1;
	}
	hipDeviceProp_t deviceProp;
	CUDA_CHECKED_CALL(hipGetDeviceProperties(&deviceProp, 0));
	cout << "Using device: " << deviceProp.name << endl;

	size_t n = (size_t)strtoull(argv[1], NULL, 0);
	if (n % BLOCK_SIZE)
	{
		cerr << "For simplisity, we require n (" << n <<
			") to be exact multiplier of BLOCK_SIZE (" <<
			std::to_string(static_cast<long long>(BLOCK_SIZE)) << ")" << endl;
		return -1;
	}

	float *A1 = new float[n * n], *A2 = new float[n * n], *A3 = new float[n * n];
	float *B1 = new float[n * n], *B2 = new float[n * n], *B3 = new float[n * n];
	float *C1 = new float[n * n], *C2 = new float[n * n], *C3 = new float[n * n];

	// Generate random input matrices.
	double dinvrandmax = (double)1.0 / RAND_MAX;
	for (size_t i = 0, length = n * n; i < length; i++)
	{
		A1[i] = rand() * dinvrandmax; A2[i] = A1[i]; A3[i] = A1[i];
		B1[i] = rand() * dinvrandmax; B2[i] = B1[i]; B3[i] = B1[i];
	}
	memset(C1, 0, sizeof(float) * n * n);
	memset(C2, 0, sizeof(float) * n * n);
	memset(C3, 0, sizeof(float) * n * n);

	float time;
	Matmul(A1, B1, C1, n, MatmulVersion::CUBLAS, &time);
	cout << "CUBLAS      version completed in " << time << " sec" << endl;

	Matmul(A2, B2, C2, n, MatmulVersion::CUDA, &time);
	cout << "CUDA        version completed in " << time << " sec" << endl;

	Matmul(A3, B3, C3, n, MatmulVersion::WHIPPLETREE, &time);
	cout << "WHIPPLETREE version completed in " << time << " sec" << endl;

	// Compare C1 and C2 results.
	int status = 0;
	for (int j = 0; j < n; j++)
	{
		for (int i = 0; i < n; i++)
		{
			float c1 = C1[i + j * n];
			float c2 = C2[i * n + j];
			if (fabsf(c1 - c2) > 0.1f)
			{
				cerr << "Mismatching C2 result @ [" << i << "][" << j << "]: " << c1 << " != " << c2 << endl;
				status = -1;
				break;
			}
		}
		if (status == -1) break;
	}

	// Compare C1 and C3 results.
	for (int j = 0; j < n; j++)
	{
		for (int i = 0; i < n; i++)
		{
			float c1 = C1[i + j * n];
			float c3 = C3[i * n + j];
			if (fabsf(c1 - c3) > 0.1f)
			{
				cerr << "Mismatching C3 result @ [" << i << "][" << j << "]: " << c1 << " != " << c3 << endl;
				status = -1;
				break;
			}
		}
		if (status == -1) break;
	}

	delete[] A1; delete[] A2; delete[] A3;
	delete[] B1; delete[] B2; delete[] B3;
	delete[] C1; delete[] C2; delete[] C3;

	return status;
}

